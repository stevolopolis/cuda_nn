#include "matrix.hpp"

Matrix::Matrix(size_t x_dim, size_t y_dim) :
    shape(x_dim, y_dim), 
    data_device(nullptr),
    data_host(nullptr),
    device_allocated(false),
    host_allocated(false) 
{}


Matrix::Matrix(Shape shape) :
    Matrix(shape.x, shape.y)
{}


void Matrix::allocateHostMem() {
    if (!host_allocated) {
        data_host = std::shared_ptr<float>(new float[shape.x * shape.y], [&](float* ptr) { delete[] ptr;});
        host_allocated = true;
    }
}


void Matrix::allocateDeviceMem() {
    if (!device_allocated) {
        float *device_memory = nullptr;
        hipMalloc(&device_memory, shape.x * shape.y * sizeof(float));
        data_device = std::shared_ptr<float>(device_memory, [&](float *ptr) { hipFree(ptr); });

        device_allocated = true;
    }
}


void Matrix::allocateMem() {
    allocateHostMem();
    allocateDeviceMem();
}


void Matrix::allocateMemIfNotAllocated(Shape shape) {
    if (!host_allocated && !device_allocated) {
        this->shape = shape;
        allocateMem();
    }
}


void Matrix::copyHostToDevice() {
    if (host_allocated && device_allocated) {
        hipMemcpy(data_device.get(), data_host.get(), shape.x * shape.y * sizeof(float), hipMemcpyHostToDevice);
    }
}


void Matrix::copyDeviceToHost() {
    if (host_allocated && device_allocated) {
        hipMemcpy(data_host.get(), data_device.get(), shape.x * shape.y * sizeof(float), hipMemcpyDeviceToHost);
    }
}